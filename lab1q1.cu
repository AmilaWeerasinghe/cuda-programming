#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>

#define SIZE 10

__global__  void addVector(float *vectorAns_cuda, float *vectorA_cuda );

int main(){

    //arrays in main memory
    float vectorA[SIZE];
    float vectorAns[SIZE/2];

    //generate some values (use randn())
    /*for(int i=0;i<SIZE;i++){
        vectorA[i]=rand();
    }*/
    //known vector for test answer 
    for(i=0;i<SIZE;i++){
		vectorA[i]=i;
	}

    //pointers for arrays to be put on cuda memory
    float *vectorA_cuda;
    float *vectorAns_cuda;

    //for error checking
    hipError_t code;
    
    //allocate memory in cuda device
    hipMalloc((void **)&vectorA_cuda,sizeof(float)*SIZE);
	code = hipGetLastError();
    assert (code == hipSuccess);
    
    hipMalloc((void **)&vectorAns_cuda,sizeof(float)*(SIZE/2));
	code = hipGetLastError();
    assert (code == hipSuccess);
    
    //copy contents from main memory to cuda device memory
    hipMemcpy(vectorA_cuda,vectorA,sizeof(float)*SIZE,hipMemcpyHostToDevice);
	code = hipGetLastError();
    assert (code == hipSuccess);
    
    //call the cuda kernel
    addVector<<<1,SIZE>>>(vectorAns_cuda, vectorA_cuda);
    hipDeviceSynchronize();
	code = hipGetLastError();
    assert (code == hipSuccess);
    
    //copy back the results from cuda memory to main memory
    hipMemcpy(vectorAns,vectorAns_cuda,sizeof(float)*(SIZE/2),hipMemcpyDeviceToHost);
    code = hipGetLastError();
    assert (code == hipSuccess);
    

    //free memory
    hipFree(vectorA_cuda);
    hipFree(vectorAns_cuda);


    printf("Answer is : ");

    for(i=0;i<(SIZE/2);i++){
		printf("%d ",vectorAns[i]);
	}
	
	return 0;


}


__global__  void addVector(float *vectorAns_cuda, float *vectorA_cuda ){
	
    int tid=threadIdx.x;
    /*if(tid==0){
        //the first element
        vectorAns_cuda[tid/2]=vectorA_cuda[tid]+vectorA_cuda[tid+1];
    }
    else if(tid%2==0){
        //other indexes than 0
    vectorAns_cuda[tid-1]=vectorA_cuda[tid]+vectorA_cuda[tid+1];
}*/
    if(tid%2==0){
        vectorAns_cuda[tid/2]=(vectorA_cuda[tid]+vectorA_cuda[tid+1])/2;
    }

  

}